/**
 *  The MIT License (MIT)
 *
 *  Copyright (c) 2015 Kyle Hollins Wray, University of Massachusetts
 *
 *  Permission is hereby granted, free of charge, to any person obtaining a copy of
 *  this software and associated documentation files (the "Software"), to deal in
 *  the Software without restriction, including without limitation the rights to
 *  use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies of
 *  the Software, and to permit persons to whom the Software is furnished to do so,
 *  subject to the following conditions:
 *
 *  The above copyright notice and this permission notice shall be included in all
 *  copies or substantial portions of the Software.
 *
 *  THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 *  IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS
 *  FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR
 *  COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER
 *  IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN
 *  CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 */


#include "mdp_model_gpu.h"
#include "error_codes.h"

#include <stdio.h>


int mdp_initialize_successors_gpu(MDP *mdp)
{
    // Ensure the data is valid.
    if (mdp->n == 0 || mdp->m == 0 || mdp->ns == 0 || mdp->S == nullptr) {
        fprintf(stderr, "Error[mdp_initialize_successors_gpu]: %s\n", "Invalid input.");
        return NOVA_ERROR_INVALID_DATA;
    }

    // Allocate the memory on the device.
    if (hipMalloc(&mdp->d_S, mdp->n * mdp->m * mdp->ns * sizeof(int)) != hipSuccess) {
        fprintf(stderr, "Error[mdp_initialize_successors_gpu]: %s\n",
                "Failed to allocate device-side memory for the successor states.");
        return NOVA_ERROR_DEVICE_MALLOC;
    }

    // Copy the data from the host to the device.
    if (hipMemcpy(mdp->d_S, mdp->S, mdp->n * mdp->m * mdp->ns * sizeof(int),
                    hipMemcpyHostToDevice) != hipSuccess) {
        fprintf(stderr, "Error[mdp_initialize_successors_gpu]: %s\n",
                "Failed to copy memory from host to device for the successor states.");
        return NOVA_ERROR_MEMCPY_TO_DEVICE;
    }

    return NOVA_SUCCESS;
}


int mdp_uninitialize_successors_gpu(MDP *mdp)
{
    if (mdp->d_S != nullptr) {
        if (hipFree(mdp->d_S) != hipSuccess) {
            fprintf(stderr, "Error[mdp_uninitialize_successors_gpu]: %s\n",
                    "Failed to allocate device-side memory for the successor states.");
            return NOVA_ERROR_DEVICE_FREE;
        }
    }
    mdp->d_S = nullptr;

    return NOVA_SUCCESS;
}


int mdp_initialize_state_transitions_gpu(MDP *mdp)
{
    // Ensure the data is valid.
    if (mdp->n == 0 || mdp->m == 0 || mdp->ns == 0 || mdp->T == nullptr) {
        fprintf(stderr, "Error[mdp_initialize_state_transitions_gpu]: %s\n", "Invalid input.");
        return NOVA_ERROR_INVALID_DATA;
    }

    // Allocate the memory on the device.
    if (hipMalloc(&mdp->d_T, mdp->n * mdp->m * mdp->ns * sizeof(float)) != hipSuccess) {
        fprintf(stderr, "Error[mdp_initialize_state_transitions_gpu]: %s\n",
                "Failed to allocate device-side memory for the state transitions.");
        return NOVA_ERROR_DEVICE_MALLOC;
    }

    // Copy the data from the host to the device.
    if (hipMemcpy(mdp->d_T, mdp->T, mdp->n * mdp->m * mdp->ns * sizeof(float), hipMemcpyHostToDevice) != hipSuccess) {
        fprintf(stderr, "Error[nova_mdp_pbvi_initialize_state_transitions]: %s\n",
                "Failed to copy memory from host to device for the state transitions.");
        return NOVA_ERROR_MEMCPY_TO_DEVICE;
    }

    return NOVA_SUCCESS;
}


int mdp_uninitialize_state_transitions_gpu(MDP *mdp)
{
    if (mdp->d_T != nullptr) {
        if (hipFree(mdp->d_T) != hipSuccess) {
            fprintf(stderr, "Error[mdp_uninitialize_state_transitions_gpu]: %s\n",
                    "Failed to allocate device-side memory for the state transitions.");
            return NOVA_ERROR_DEVICE_FREE;
        }
    }
    mdp->d_T = nullptr;

    return NOVA_SUCCESS;
}


int mdp_initialize_rewards_gpu(MDP *mdp)
{
    // Ensure the data is valid.
    if (mdp->n == 0 || mdp->m == 0 || mdp->R == nullptr) {
        fprintf(stderr, "Error[mdp_initialize_rewards_gpu]: %s\n", "Invalid input.");
        return NOVA_ERROR_INVALID_DATA;
    }

    // Allocate the memory on the device.
    if (hipMalloc(&mdp->d_R, mdp->n * mdp->m * sizeof(float)) != hipSuccess) {
        fprintf(stderr, "Error[mdp_initialize_rewards_gpu]: %s\n",
                "Failed to allocate device-side memory for the rewards.");
        return NOVA_ERROR_DEVICE_MALLOC;
    }

    // Copy the data from the host to the device.
    if (hipMemcpy(mdp->d_R, mdp->R, mdp->n * mdp->m * sizeof(float), hipMemcpyHostToDevice) != hipSuccess) {
        fprintf(stderr, "Error[mdp_initialize_rewards_gpu]: %s\n",
                "Failed to copy memory from host to device for the rewards.");
        return NOVA_ERROR_MEMCPY_TO_DEVICE;
    }

    return NOVA_SUCCESS;
}


int mdp_uninitialize_rewards_gpu(MDP *mdp)
{
    if (mdp->d_R != nullptr) {
        if (hipFree(mdp->d_R) != hipSuccess) {
            fprintf(stderr, "Error[mdp_uninitialize_rewards_gpu]: %s\n",
                    "Failed to allocate device-side memory for the rewards.");
            return NOVA_ERROR_DEVICE_FREE;
        }
    }
    mdp->d_R = nullptr;

    return NOVA_SUCCESS;
}

